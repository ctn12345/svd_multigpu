#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include<omp.h>

#include "matrix_generate.hpp"
#include "large_matrix_svd.cu"
#include "small_matrix_svd.cu"

// #include "cusolver_svd.cu"

using namespace std;
using namespace std::chrono;
// fig 14 a
// 100x512x512 speedup over cusolver(CUDA platform)
void test17(){
    int gpu0=0,gpu1=1;
    int batch = 1;
    int height = 4096;
    int width = 4096;
    int th=0, tw=0;
    // int shape[3] = {batch, height, width};
    int minmn = height > width/2 ? width/2 : height;

    double* host_A = (double*)malloc(sizeof(double) * height * width);
    string matrix_path1 = "./data/generated_matrixes/A_h" + to_string(height) + "_w" + to_string(width)+ ".txt";

    // read in host A
    FILE* A_fp = fopen(matrix_path1.data(), "r");
    if(A_fp==NULL){
        generate_matrix(height, width);
        A_fp = fopen(matrix_path1.data(), "r");
        if(A_fp==NULL){
            printf("open file falied\n");
            return ;
        }
    }
    for(int i=0; i < height*width; i++){
        fscanf(A_fp, "%lf", &host_A[i]);
    }

    fclose(A_fp);

    // steady_clock::time_point t1 = steady_clock::now();
    
    // double *dev_A;
    // hipMalloc((void **)&dev_A, sizeof(double) * height * width * batch);
    // for(int i=0; i<batch; i++){
    //     hipMemcpy(dev_A + height*width*i, host_A, sizeof(double) * height * width, hipMemcpyHostToDevice);
    // }
    // double *dev_U, *dev_V, *dev_diag;
    // hipMalloc((void **)&dev_diag, sizeof(double) * minmn * batch);
    // hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
    // hipMalloc((void **)&dev_V, sizeof(double) * width * width * batch);
    tw = 32;
    th = 32;
    int k = tw/2;
    int slice = th;
    int width_perdevice=width/2;

    // printf("input matrix shape: %d × %d × %d, tile shape: %d × %d\n", batch, height0, width0, th, tw);

// prams
// definition 1---gpu1
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    height = q * slice;
    int sliceNum = q;
    
    double* dev_A;  // fixed A
    double* dev_V;
    double* dev_V0;
    double* dev_U;
	int* dev_roundRobin; 
    
    double* dev_jointG;
    double* dev_Aij;

    double* dev_AiAi;   
    double* dev_AiAj;
    double* dev_AjAj;
    int* dev_pairsOfEVD;
    unsigned* host_allpass;
    unsigned* host_pass;
    unsigned* dev_allpass;
    unsigned* dev_pass;
    double *value;
    double* dev_swap_data;
    unsigned int *arr1;
    unsigned int *arr2;
    unsigned int *pairs;
    double *dev_norm;
    unsigned int *dev_order;
    double* host_Fnorm; 
    double* dev_tempFnorm;
    double* dev_Fnorm;
    double* dev_diag;
#pragma endregion
// definition 2 gpu
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p1 = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q1 = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    int height1 = q * slice;
    int sliceNum1 = q;
    
    double* dev_A_1;  // fixed A
    double* dev_V_1;
    double* dev_U_1;
    double* dev_V1;
	int* dev_roundRobin_1; 
    
    double* dev_jointG_1;
    double* dev_Aij_1;

    double* dev_AiAi_1;   
    double* dev_AiAj_1;
    double* dev_AjAj_1;
    int* dev_pairsOfEVD_1;
    unsigned* host_allpass_1;
    unsigned* host_pass_1;
    unsigned* dev_allpass_1;
    unsigned* dev_pass_1;
    double *value_1;
    double* dev_swap_data_1;
    unsigned int *arr1_1;
    unsigned int *arr2_1;
    unsigned int *pairs_1;
    double *dev_norm_1;
    unsigned int *dev_order_1;
    double* host_Fnorm_1; 
    double* dev_tempFnorm_1;
    double* dev_Fnorm_1;
    double* dev_diag_1;
#pragma endregion

// memory allocate


#pragma region
    hipSetDevice(gpu0);
    hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
    hipMalloc((void **)&dev_A, sizeof(double) * height * width_perdevice * batch);
    hipMalloc((void **)&dev_V, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_V0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_diag,sizeof(double) * minmn);
    // dev_U = dev_U0;
    // dev_diag = dev_diag0;

    
    // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

    hipMalloc((void **)&dev_roundRobin, sizeof(int) * (2 * k - 1) * 2 * k);

    hipMalloc((void **)&dev_jointG, sizeof(double) * 2*k * 2*k * p*batch);
    hipMalloc((void **)&dev_Aij, sizeof(double) * height * 2*k * p*batch);

    hipMalloc((void **)&dev_AiAi, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_AiAj, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_AjAj, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_pairsOfEVD, sizeof(int) * 2 * p * batch);
    // hipMalloc((void **)&dev_swap_data,sizeof(double)*p*height*k);

    host_allpass = (unsigned *)malloc(sizeof(unsigned) * batch);
    host_pass = (unsigned *)malloc(sizeof(unsigned) * p * batch);
    hipMalloc((void **)&dev_allpass, sizeof(unsigned) * batch);
    hipMalloc((void **)&dev_pass, sizeof(unsigned) * p * batch);

    hipMalloc((void **)&dev_norm, sizeof(double) * 2 * p * batch);
    hipMalloc((void **)&dev_order, sizeof(unsigned int) * 2 * p * batch);
    host_Fnorm = (double *)malloc(sizeof(double) * batch);
    hipMalloc((void **)&dev_tempFnorm, sizeof(double) * 2 * p * batch);
    hipMalloc((void **)&dev_Fnorm, sizeof(double) * batch);

#pragma endregion
#pragma region
    hipSetDevice(gpu1);
    hipMalloc((void **)&dev_diag_1,sizeof(double) * minmn);
    hipMalloc((void **)&dev_A_1, sizeof(double) * height * width_perdevice * batch);
    hipMalloc((void **)&dev_V_1, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_V1, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_U_1, sizeof(double) * height * height * batch);
    // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

    hipMalloc((void **)&dev_roundRobin_1, sizeof(int) * (2 * k - 1) * 2 * k);

    hipMalloc((void **)&dev_jointG_1, sizeof(double) * 2*k * 2*k * p*batch);
    hipMalloc((void **)&dev_Aij_1, sizeof(double) * height * 2*k * p*batch);

    hipMalloc((void **)&dev_AiAi_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_AiAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_AjAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_pairsOfEVD_1, sizeof(int) * 2 * p1 * batch);
    // hipMalloc((void **)&dev_swap_data_1,sizeof(double)*p1*height*k);

    host_allpass_1 = (unsigned *)malloc(sizeof(unsigned) * batch);
    host_pass_1 = (unsigned *)malloc(sizeof(unsigned) * p1 * batch);
    hipMalloc((void **)&dev_allpass_1, sizeof(unsigned) * batch);
    hipMalloc((void **)&dev_pass_1, sizeof(unsigned) * p1 * batch);

    hipMalloc((void **)&dev_norm_1, sizeof(double) * 2 * p1 * batch);
    hipMalloc((void **)&dev_order_1, sizeof(unsigned int) * 2 * p1 * batch);
    host_Fnorm_1 = (double *)malloc(sizeof(double) * batch);
    hipMalloc((void **)&dev_tempFnorm_1, sizeof(double) * 2 * p1 * batch);
    hipMalloc((void **)&dev_Fnorm_1, sizeof(double) * batch);

#pragma endregion

// preset before svd  
#pragma region
    hipSetDevice(gpu0);
    hipMemset(dev_V, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMemset(dev_U, 0, sizeof(double) * height * height * batch);
    // hipMemset(dev_V0, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMemset(dev_pairsOfEVD, 0, sizeof(int) * 2 * p * batch); 
    memset(host_pass, 0, sizeof(unsigned) * p * batch);
    hipMemset(dev_pass, 0, sizeof(unsigned) * p * batch);
    int shape[3]={batch,height,width_perdevice};
    double* host_A1,* host_A2;
    host_A1 = (double*)malloc(sizeof(double)*width_perdevice*height);
    host_A2 = (double*)malloc(sizeof(double)*width_perdevice*height);
    double test_result[4] = {0, 1.0, 1.0, 1.0}; // 0:tag, 1:time
    test_result[0] = 2.0;
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipSetDevice(gpu1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);
    hipSetDevice(gpu0);
    hipMemcpyAsync(dev_A,host_A,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice,stream1);
    hipSetDevice(gpu1);
    hipMemcpyAsync(dev_A_1,host_A+height*width_perdevice,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice,stream2);
    // hipMemcpy(host_A2,dev_A_1,sizeof(double)*p1*k*height,hipMemcpyDeviceToHost);
    // printf("host_A1 %d\n",i);
    // for(int j = 0;j < p1*k*height;++j){
    //     printf("%lf ",host_A2[j]);
    // }
    // printf("\n");

    // double* dev_test_A = (double*)malloc(sizeof(double)*2*p*k*height);
    // hipMemcpy(dev_test_A,host_A,sizeof(double)*2*p*k*height,hipMemcpyHostToDevice);
    // int shape1[3] = 
    // our svd

    double* swap_data_1 = (double*)malloc(sizeof(double)*p*height*k);
    double* swap_data_2 = (double*)malloc(sizeof(double)*p*height*k);
    clock_t start1,end;
    start1 = clock();
    double t1=0,t2=0,t3=0;
    clock_t begin1,end1;
    begin1 = clock();
    hipSetDevice(gpu0);
    dim3 dimGrid0(1, 1, 1);
    dim3 dimBlock0(32, 32, 1);
    generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream1>>>(dev_roundRobin, 2*k);
    hipSetDevice(gpu1);
    generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream2>>>(dev_roundRobin_1, 2*k);
    end1 = clock();
    t3 += (double)(end1-begin1)/CLOCKS_PER_SEC;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    omp_set_num_threads(2);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);  // 等待 stream1 完
    for(int i = 0;i < 4;++i){        
        float milliseconds = 0.0f;
        // 记录 SVD 计算时间
        hipEventRecord(start, stream1);
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            if(gpuid == 0){
                printf("enter thread1 %d \n",gpuid);
                svd_large_matrix_1(gpu0, stream1, false, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm);
            }
            else{
                printf("enter thread1 %d \n",gpuid);
                svd_large_matrix_1(gpu1, stream2, false, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1);
            }
        }
        // hipSetDevice(gpu0);
        hipSetDevice(gpu0);
        hipStreamSynchronize(stream1);  // 等待 stream1 完
        // hipMemcpy(host_A1,dev_A+ p * k * height,sizeof(double)*p*k*height,hipMemcpyDeviceToHost);
        // for(int y = 0;y < 5;++y){
        //     printf("%lf ",host_A1[y]);
        // }
        // printf("\n");
        hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);

        hipSetDevice(gpu1);
        hipStreamSynchronize(stream2);  // 等待 stream1 完
        hipMemcpyAsync(swap_data_2, dev_A_1 + p1 * k * height, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);

        hipStreamSynchronize(stream2);  // 等待 stream2 完
        hipSetDevice(gpu0);
        
        hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
        hipStreamSynchronize(stream1);  // 等待 stream1 完
        // for(int y = 0;y < 5;++y){
        //     printf("%lf ",swap_data_1[y]);
        // }
        hipSetDevice(gpu1);
        hipMemcpyAsync(dev_A_1 + p1 * k * height, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
        hipStreamSynchronize(stream2);  // 等待 stream2 完

        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        t2 += milliseconds / 1000.0;  // 转换为秒

        // 第二轮 SVD 计算
        hipEventRecord(start, stream1);
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            if(gpuid == 0){
                svd_large_matrix_1(gpu0, stream1, false, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm);
            }
            else{
                svd_large_matrix_1(gpu1, stream2, false, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1);
            }
        }
        // hipEventRecord(stop, stream1);
        // hipEventSynchronize(stop);
        // hipEventElapsedTime(&milliseconds, start, stop);
        // t1 += milliseconds / 1000.0;  // 转换为秒

        // 交换数据
        hipEventRecord(start, stream1);
        hipSetDevice(gpu0);
        hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);

        hipSetDevice(gpu1);
        hipMemcpyAsync(swap_data_2, dev_A_1, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);

        hipStreamSynchronize(stream2);
        hipSetDevice(gpu0);
        hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
        hipStreamSynchronize(stream1);

        hipSetDevice(gpu1);
        hipMemcpyAsync(dev_A_1, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
        hipStreamSynchronize(stream2);

        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        t2 += milliseconds / 1000.0;  // 转换为秒

        bool flag = (i == 3);

        // 第三轮 SVD 计算
        hipEventRecord(start, stream1);
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            if(gpuid == 0){
                svd_large_matrix_1(gpu0, stream1, flag, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm);
            }
            else{
                svd_large_matrix_1(gpu1, stream2, flag, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1);
            }
        }
        hipEventRecord(stop, stream1);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        t1 += milliseconds / 1000.0;  // 转换为秒

        // 只有 i != 3 时进行数据交换
        if (i != 3) {
            hipEventRecord(start, stream1);
            hipSetDevice(gpu0);
            hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);

            hipSetDevice(gpu1);
            hipMemcpyAsync(swap_data_2, dev_A_1 + p1 * k * height, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);
            hipStreamSynchronize(stream2);

            hipSetDevice(gpu0);
            hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
            hipStreamSynchronize(stream1);

            hipSetDevice(gpu1);
            hipMemcpyAsync(dev_A_1 + p1 * k * height, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
            hipStreamSynchronize(stream2);

            hipEventRecord(stop, stream1);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            t2 += milliseconds / 1000.0;  // 转换为秒
            }
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    end1 = clock();
    printf("it costs %lfs",(double)(end1-begin1)/CLOCKS_PER_SEC);
    
    double* host_diag = (double*)malloc(sizeof(double)*minmn*batch);
    double* host_diag1 = (double*)malloc(sizeof(double)*minmn*batch);
    hipSetDevice(gpu0);
    hipMemcpy(host_diag,dev_diag,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    hipSetDevice(gpu1);
    hipMemcpy(host_diag1,dev_diag_1,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    FILE* file2 = fopen("dev_diag.txt","w");
    for(int f = 0;f < minmn;++f){
        fprintf(file2,"%lf %lf ",host_diag[f],host_diag1[f]);
    }
    
    printf("matrix:%d×%d×%d, speedup over cusolver: %lf/%lf = %lf\n", batch, height, width, test_result[2], test_result[1], test_result[2]/test_result[1]); 

    free(host_A);
    hipFree(dev_A);
    hipFree(dev_U);
    hipFree(dev_V);
    hipFree(dev_diag);
    hipDeviceReset();
}

int main(int argc, char* argv[]){
    test17();
}