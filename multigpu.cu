#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include<omp.h>
#include <nvToolsExt.h>

#include "matrix_generate.hpp"
#include "large_matrix_svd.cu"
#include "small_matrix_svd.cu"

// #include "cusolver_svd.cu"

using namespace std;
using namespace std::chrono;
// fig 14 a
// 100x512x512 speedup over cusolver(CUDA platform)
void test17(){
    int gpu0=0,gpu1=1;
    int batch = 1;
    int height = 8192;
    int width = 8192;
    int th=0, tw=0;
    // int shape[3] = {batch, height, width};
    int minmn = height > width/2 ? width/2 : height;

    double* host_A = (double*)malloc(sizeof(double) * height * width);
    string matrix_path1 = "./data/generated_matrixes/A_h" + to_string(height) + "_w" + to_string(width)+ ".txt";

    // read in host A
    FILE* A_fp = fopen(matrix_path1.data(), "r");
    if(A_fp==NULL){
        generate_matrix(height, width);
        A_fp = fopen(matrix_path1.data(), "r");
        if(A_fp==NULL){
            printf("open file falied\n");
            return ;
        }
    }
    for(int i=0; i < height*width; i++){
        fscanf(A_fp, "%lf", &host_A[i]);
    }

    fclose(A_fp);

    // steady_clock::time_point t1 = steady_clock::now();
    
    // double *dev_A;
    // hipMalloc((void **)&dev_A, sizeof(double) * height * width * batch);
    // for(int i=0; i<batch; i++){
    //     hipMemcpy(dev_A + height*width*i, host_A, sizeof(double) * height * width, hipMemcpyHostToDevice);
    // }
    // double *dev_U, *dev_V, *dev_diag;
    // hipMalloc((void **)&dev_diag, sizeof(double) * minmn * batch);
    // hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
    // hipMalloc((void **)&dev_V, sizeof(double) * width * width * batch);
    tw = 32;
    th = 32;
    int k = tw/2;
    int slice = th;
    int width_perdevice=width/2;

    // printf("input matrix shape: %d × %d × %d, tile shape: %d × %d\n", batch, height0, width0, th, tw);

// prams
// definition 1---gpu1
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    height = q * slice;
    int sliceNum = q;
    
    double* dev_A;  // fixed A
    double* dev_V;
    double* dev_V0;
    double* dev_U;
	int* dev_roundRobin; 
    
    double* dev_jointG;
    double* dev_Aij;

    double* dev_AiAi;   
    double* dev_AiAj;
    double* dev_AjAj;
    int* dev_pairsOfEVD;
    unsigned* host_allpass;
    unsigned* host_pass;
    unsigned* dev_allpass;
    unsigned* dev_pass;
    double *value;
    double* dev_swap_data;
    unsigned int *arr1;
    unsigned int *arr2;
    unsigned int *pairs;
    double *dev_norm;
    unsigned int *dev_order;
    double* host_Fnorm; 
    double* dev_tempFnorm;
    double* dev_Fnorm;
    double* dev_diag;
#pragma endregion
// definition 2 gpu
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p1 = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q1 = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    int height1 = q * slice;
    int sliceNum1 = q;
    
    double* dev_A_1;  // fixed A
    double* dev_V_1;
    double* dev_U_1;
    double* dev_V1;
	int* dev_roundRobin_1; 
    
    double* dev_jointG_1;
    double* dev_Aij_1;

    double* dev_AiAi_1;   
    double* dev_AiAj_1;
    double* dev_AjAj_1;
    int* dev_pairsOfEVD_1;
    unsigned* host_allpass_1;
    unsigned* host_pass_1;
    unsigned* dev_allpass_1;
    unsigned* dev_pass_1;
    double *value_1;
    double* dev_swap_data_1;
    unsigned int *arr1_1;
    unsigned int *arr2_1;
    unsigned int *pairs_1;
    double *dev_norm_1;
    unsigned int *dev_order_1;
    double* host_Fnorm_1; 
    double* dev_tempFnorm_1;
    double* dev_Fnorm_1;
    double* dev_diag_1;
#pragma endregion

// memory allocate


#pragma region
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipSetDevice(gpu1);
    hipStream_t stream2;
    hipStreamCreate(&stream2);
    omp_set_num_threads(2);
#pragma endregion


#pragma region
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        if(gpuid == 0){
            hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
            hipMalloc((void **)&dev_A, sizeof(double) * height * width_perdevice * batch);
            hipMalloc((void **)&dev_V, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMalloc((void **)&dev_V0, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMalloc((void **)&dev_diag,sizeof(double) * minmn);
            // dev_U = dev_U0;
            // dev_diag = dev_diag0;            
            // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

            hipMalloc((void **)&dev_roundRobin, sizeof(int) * (2 * k - 1) * 2 * k);

            hipMalloc((void **)&dev_jointG, sizeof(double) * 2*k * 2*k * p*batch);
            hipMalloc((void **)&dev_Aij, sizeof(double) * height * 2*k * p*batch);

            hipMalloc((void **)&dev_AiAi, sizeof(double) * k * k * sliceNum * p * batch);
            hipMalloc((void **)&dev_AiAj, sizeof(double) * k * k * sliceNum * p * batch);
            hipMalloc((void **)&dev_AjAj, sizeof(double) * k * k * sliceNum * p * batch);
            hipMalloc((void **)&dev_pairsOfEVD, sizeof(int) * 2 * p * batch);
            // hipMalloc((void **)&dev_swap_data,sizeof(double)*p*height*k);

            host_allpass = (unsigned *)malloc(sizeof(unsigned) * batch);
            host_pass = (unsigned *)malloc(sizeof(unsigned) * p * batch);
            hipMalloc((void **)&dev_allpass, sizeof(unsigned) * batch);
            hipMalloc((void **)&dev_pass, sizeof(unsigned) * p * batch);

            hipMalloc((void **)&dev_norm, sizeof(double) * 2 * p * batch);
            hipMalloc((void **)&dev_order, sizeof(unsigned int) * 2 * p * batch);
            host_Fnorm = (double *)malloc(sizeof(double) * batch);
            hipMalloc((void **)&dev_tempFnorm, sizeof(double) * 2 * p * batch);
            hipMalloc((void **)&dev_Fnorm, sizeof(double) * batch);

            hipMemset(dev_V, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMemset(dev_U, 0, sizeof(double) * height * height * batch);
            // hipMemset(dev_V0, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMemset(dev_pairsOfEVD, 0, sizeof(int) * 2 * p * batch); 
            memset(host_pass, 0, sizeof(unsigned) * p * batch);
            hipMemset(dev_pass, 0, sizeof(unsigned) * p * batch);
            hipMemcpyAsync(dev_A,host_A,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice,stream1);
        }
        else{
            hipMalloc((void **)&dev_diag_1,sizeof(double) * minmn);
            hipMalloc((void **)&dev_A_1, sizeof(double) * height * width_perdevice * batch);
            hipMalloc((void **)&dev_V_1, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMalloc((void **)&dev_V1, sizeof(double) * width_perdevice * width_perdevice * batch);
            hipMalloc((void **)&dev_U_1, sizeof(double) * height * height * batch);
            // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

            hipMalloc((void **)&dev_roundRobin_1, sizeof(int) * (2 * k - 1) * 2 * k);

            hipMalloc((void **)&dev_jointG_1, sizeof(double) * 2*k * 2*k * p*batch);
            hipMalloc((void **)&dev_Aij_1, sizeof(double) * height * 2*k * p*batch);

            hipMalloc((void **)&dev_AiAi_1, sizeof(double) * k * k * sliceNum * p1 * batch);
            hipMalloc((void **)&dev_AiAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
            hipMalloc((void **)&dev_AjAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
            hipMalloc((void **)&dev_pairsOfEVD_1, sizeof(int) * 2 * p1 * batch);
            // hipMalloc((void **)&dev_swap_data_1,sizeof(double)*p1*height*k);

            host_allpass_1 = (unsigned *)malloc(sizeof(unsigned) * batch);
            host_pass_1 = (unsigned *)malloc(sizeof(unsigned) * p1 * batch);
            hipMalloc((void **)&dev_allpass_1, sizeof(unsigned) * batch);
            hipMalloc((void **)&dev_pass_1, sizeof(unsigned) * p1 * batch);

            hipMalloc((void **)&dev_norm_1, sizeof(double) * 2 * p1 * batch);
            hipMalloc((void **)&dev_order_1, sizeof(unsigned int) * 2 * p1 * batch);
            host_Fnorm_1 = (double *)malloc(sizeof(double) * batch);
            hipMalloc((void **)&dev_tempFnorm_1, sizeof(double) * 2 * p1 * batch);
            hipMalloc((void **)&dev_Fnorm_1, sizeof(double) * batch);
            hipMemcpyAsync(dev_A_1,host_A+height*width_perdevice,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice,stream2);
        }
    }
#pragma endregion

// preset before svd  
#pragma region
    hipSetDevice(gpu0);
    
    int shape[3]={batch,height,width_perdevice};
    double* host_A1,* host_A2;
    host_A1 = (double*)malloc(sizeof(double)*width_perdevice*height);
    host_A2 = (double*)malloc(sizeof(double)*width_perdevice*height);
    double test_result[4] = {0, 1.0, 1.0, 1.0}; // 0:tag, 1:time
    test_result[0] = 2.0;
    // omp thread

    double* swap_data_1 = (double*)malloc(sizeof(double)*p*height*k);
    double* swap_data_2 = (double*)malloc(sizeof(double)*p*height*k);
    clock_t start1,end;
    start1 = clock();
    double t1=0,t2=0,t3=0;
    clock_t begin1,end1;
    begin1 = clock();
    hipSetDevice(gpu0);
    dim3 dimGrid0(1, 1, 1);
    dim3 dimBlock0(32, 32, 1);
    
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        if (gpuid == 0)
            generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream1>>>(dev_roundRobin, 2*k);
        else
            generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream2>>>(dev_roundRobin_1, 2*k);
    }
    // generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream1>>>(dev_roundRobin, 2*k);
    // hipSetDevice(gpu1);
    // generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream2>>>(dev_roundRobin_1, 2*k);
    // end1 = clock();
    // t3 += (double)(end1-begin1)/CLOCKS_PER_SEC;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
   
    
    double test_tag=0;
    int i = 0;
    while(test_tag < 1){    
        test_tag=4;    
        nvtxMarkA("Start SVD Computation"); // 这里会在 Nsight Systems 里显示一个标记
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            // printf("thread %d \n",gpuid);
            hipSetDevice(gpuid);
            if(gpuid == 0){
                if(i != 0)
                    hipStreamSynchronize(stream2);  // 等待 stream1 完
                // printf("thread %d \n",gpuid);
                svd_large_matrix_1(gpu0, stream1, false, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm,&test_tag);
                hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);
            }
            else{
                if(i != 0)
                    hipStreamSynchronize(stream1);
                // printf("thread %d ",gpuid);
                svd_large_matrix_1(gpu1, stream2, false, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1,&test_tag);
                hipMemcpyAsync(swap_data_2, dev_A_1 + p1 * k * height, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);
                // hipStreamSynchronize(stream1);  // 等待 stream1 完
                // hipMemcpyAsync(dev_A_1 + p1 * k * height, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
            }
            hipSetDevice(gpuid);
            #pragma omp barrier  // **确保两个线程都完成了数据拷贝**
            if (gpuid == 0) {
                hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
            }
            else {
                hipMemcpyAsync(dev_A_1 + p1 * k * height, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
            }
        }
        nvtxMarkA("End SVD Computation");

        // 第二轮 SVD 计算
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            hipSetDevice(gpuid);
            if(gpuid == 0){ 
                hipStreamSynchronize(stream2);  // 等待 stream2 完
                // printf("thread %d ",gpuid);
                svd_large_matrix_1(gpu0, stream1, false, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm,&test_tag);
                hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);
                // hipStreamSynchronize(stream2);
            }
            else{            
                hipStreamSynchronize(stream1);  // 等待 stream1 完
                // printf("thread %d ",gpuid);
                svd_large_matrix_1(gpu1, stream2, false, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1,&test_tag);
                hipMemcpyAsync(swap_data_2, dev_A_1, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);
                // hipStreamSynchronize(stream1);
                
            }

            #pragma omp barrier  // **确保两个线程都完成了数据拷贝**

            if (gpuid == 0) {
                hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
            }
            else {
                hipMemcpyAsync(dev_A_1, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
            }
        }


        bool flag = false;

        // printf("third\n");
        // 第三轮 SVD 计算
        hipEventRecord(start, stream1);
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            hipSetDevice(gpuid);
            if(gpuid == 0){
                hipStreamSynchronize(stream2);
                // printf("thread %d ",gpuid);
                svd_large_matrix_1(gpu0, stream1, flag, dev_A, shape, dev_diag, dev_U, dev_V, dev_V0, 
                    th, tw, dev_roundRobin, dev_jointG, dev_Aij, dev_AiAi, dev_AiAj, 
                    dev_AjAj, dev_pairsOfEVD, dev_allpass, dev_pass, dev_norm, 
                    dev_order, dev_tempFnorm, dev_Fnorm,&test_tag);
                // 只有 i != 3 时进行数据交换
                if(i != 3){
                    hipMemcpyAsync(swap_data_1, dev_A + p * k * height, sizeof(double) * p * k * height, hipMemcpyDeviceToHost, stream1);
                }  
            }
            else{
                hipStreamSynchronize(stream1);
                svd_large_matrix_1(gpu1, stream2, flag, dev_A_1, shape, dev_diag_1, dev_U_1, dev_V_1, 
                    dev_V1, th, tw, dev_roundRobin_1, dev_jointG_1, dev_Aij_1, 
                    dev_AiAi_1, dev_AiAj_1, dev_AjAj_1, dev_pairsOfEVD_1, dev_allpass_1, 
                    dev_pass_1, dev_norm_1, dev_order_1, dev_tempFnorm_1, dev_Fnorm_1,&test_tag);
                // 只有 i != 3 时进行数据交换
                if(i != 3){
                    hipMemcpyAsync(swap_data_2, dev_A_1 + p1 * k * height, sizeof(double) * p1 * k * height, hipMemcpyDeviceToHost, stream2);     
                }  
            }
           if(i != 3){
                #pragma omp barrier
                if (gpuid == 0) {
                    hipMemcpyAsync(dev_A + p * k * height, swap_data_2, sizeof(double) * p1 * k * height, hipMemcpyHostToDevice, stream1);
                }
                else {
                    hipMemcpyAsync(dev_A_1 + p1 * k * height, swap_data_1, sizeof(double) * p * k * height, hipMemcpyHostToDevice, stream2);
                }
            }  
        }
        ++i; 
    }
    #pragma omp parallel
    {
        dim3 dimGrid10(2 * p, batch, 1);
        dim3 dimBlock10(32, k, 1);
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        if(gpuid == 0){
            getUDV<<<dimGrid10, dimBlock10,0,stream1>>>(dev_A, dev_U, dev_V, dev_V0, height, width, height, width, p, height/32, dev_diag, width, k);  //&1.3
        }
        else{
            getUDV<<<dimGrid10, dimBlock10,0,stream2>>>(dev_A_1, dev_U_1, dev_V_1, dev_V1, height, width, height, width, p, height/32, dev_diag_1, width, k);  //&1.3
        }
    }
    end1 = clock();
    printf("it costs %lfs",(double)(end1-begin1)/CLOCKS_PER_SEC);
    
    double* host_diag = (double*)malloc(sizeof(double)*minmn*batch);
    double* host_diag1 = (double*)malloc(sizeof(double)*minmn*batch);
    hipSetDevice(gpu0);
    hipMemcpy(host_diag,dev_diag,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    hipSetDevice(gpu1);
    hipMemcpy(host_diag1,dev_diag_1,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    FILE* file2 = fopen("dev_diag.txt","w");
    for(int f = 0;f < minmn;++f){
        fprintf(file2,"%lf %lf ",host_diag[f],host_diag1[f]);
    }
    
    printf("matrix:%d×%d×%d, speedup over cusolver: %lf/%lf = %lf\n", batch, height, width, test_result[2], test_result[1], test_result[2]/test_result[1]); 

    free(host_A);
    hipFree(dev_A);
    hipFree(dev_U);
    hipFree(dev_V);
    hipFree(dev_diag);
    hipDeviceReset();
}

int main(int argc, char* argv[]){
    test17();
}