#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include <nvToolsExt.h>

#include "matrix_generate.hpp"
#include "large_matrix_svd.cu"
#include "small_matrix_svd.cu"

// #include "cusolver_svd.cu"

using namespace std;
using namespace std::chrono;
// fig 14 a
// 100x512x512 speedup over cusolver(CUDA platform)
void test17(){
    int gpu0 = 0,gpu1=1;
    int batch = 1;
    int height = 9216;
    int width = 9216;
    int th=0, tw=0;
    // int shape[3] = {batch, height, width};
    int minmn = height > width ? width : height;

    double* host_A = (double*)malloc(sizeof(double) * height * width);
    string matrix_path1 = "./data/generated_matrixes/A_h" + to_string(height) + "_w" + to_string(width)+ ".txt";

    // read in host A
    FILE* A_fp = fopen(matrix_path1.data(), "r");
    if(A_fp==NULL){
        generate_matrix(height, width);
        A_fp = fopen(matrix_path1.data(), "r");
        if(A_fp==NULL){
            printf("open file falied\n");
            return ;
        }
    }
    for(int i=0; i < height*width; i++){
        fscanf(A_fp, "%lf", &host_A[i]);
    }

    fclose(A_fp);

    // steady_clock::time_point t1 = steady_clock::now();
    
    // double *dev_A;
    // hipMalloc((void **)&dev_A, sizeof(double) * height * width * batch);
    // for(int i=0; i<batch; i++){
    //     hipMemcpy(dev_A + height*width*i, host_A, sizeof(double) * height * width, hipMemcpyHostToDevice);
    // }
    // double *dev_U, *dev_V, *dev_diag;
    // hipMalloc((void **)&dev_diag, sizeof(double) * minmn * batch);
    // hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
    // hipMalloc((void **)&dev_V, sizeof(double) * width * width * batch);
    tw = 32;
    th = 32;
    int k = tw/2;
    int slice = th;
    int width_perdevice=width/1;

    // printf("input matrix shape: %d × %d × %d, tile shape: %d × %d\n", batch, height0, width0, th, tw);

// prams
// definition 1---gpu1
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    height = q * slice;
    int sliceNum = q;
    
    double* dev_A;  // fixed A
    double* dev_V;
    double* dev_V0;
    double* dev_U;
	int* dev_roundRobin; 
    
    double* dev_jointG;
    double* dev_Aij;

    double* dev_AiAi;   
    double* dev_AiAj;
    double* dev_AjAj;
    int* dev_pairsOfEVD;
    unsigned* host_allpass;
    unsigned* host_pass;
    unsigned* dev_allpass;
    unsigned* dev_pass;
    double *value;
    double* dev_swap_data;
    unsigned int *arr1;
    unsigned int *arr2;
    unsigned int *pairs;
    double *dev_norm;
    unsigned int *dev_order;
    double* host_Fnorm; 
    double* dev_tempFnorm;
    double* dev_Fnorm;
    double* dev_diag;
#pragma endregion
// definition 2 gpu
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    int p1 = (width_perdevice - 1) / (2 * k) + 1; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    int q1 = (height - 1) / slice + 1;
    // int width_perdevice = p * (2 * k);
    int height1 = q * slice;
    int sliceNum1 = q;
    
    double* dev_A_1;  // fixed A
    double* dev_V_1;
    double* dev_U_1;
	int* dev_roundRobin_1; 
    
    double* dev_jointG_1;
    double* dev_Aij_1;

    double* dev_AiAi_1;   
    double* dev_AiAj_1;
    double* dev_AjAj_1;
    int* dev_pairsOfEVD_1;
    unsigned* host_allpass_1;
    unsigned* host_pass_1;
    unsigned* dev_allpass_1;
    unsigned* dev_pass_1;
    double *value_1;
    double* dev_swap_data_1;
    unsigned int *arr1_1;
    unsigned int *arr2_1;
    unsigned int *pairs_1;
    double *dev_norm_1;
    unsigned int *dev_order_1;
    double* host_Fnorm_1; 
    double* dev_tempFnorm_1;
    double* dev_Fnorm_1;
    double* dev_diag_1;
#pragma endregion

// memory allocate


#pragma region

    hipMalloc((void **)&dev_U, sizeof(double) * height * height * batch);
    hipMalloc((void **)&dev_A, sizeof(double) * height * width_perdevice * batch);
    hipMalloc((void **)&dev_V, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_V0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_diag,sizeof(double) * minmn);
    // dev_U = dev_U0;
    // dev_diag = dev_diag0;

    
    // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

    hipMalloc((void **)&dev_roundRobin, sizeof(int) * (2 * k - 1) * 2 * k);

    hipMalloc((void **)&dev_jointG, sizeof(double) * 2*k * 2*k * p*batch);
    hipMalloc((void **)&dev_Aij, sizeof(double) * height * 2*k * p*batch);

    hipMalloc((void **)&dev_AiAi, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_AiAj, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_AjAj, sizeof(double) * k * k * sliceNum * p * batch);
    hipMalloc((void **)&dev_pairsOfEVD, sizeof(int) * 2 * p * batch);
    hipMalloc((void **)&dev_swap_data,sizeof(double)*p*height*k);

    host_allpass = (unsigned *)malloc(sizeof(unsigned) * batch);
    host_pass = (unsigned *)malloc(sizeof(unsigned) * p * batch);
    hipMalloc((void **)&dev_allpass, sizeof(unsigned) * batch);
    hipMalloc((void **)&dev_pass, sizeof(unsigned) * p * batch);

    hipMalloc((void **)&dev_norm, sizeof(double) * 2 * p * batch);
    hipMalloc((void **)&dev_order, sizeof(unsigned int) * 2 * p * batch);
    host_Fnorm = (double *)malloc(sizeof(double) * batch);
    hipMalloc((void **)&dev_tempFnorm, sizeof(double) * 2 * p * batch);
    hipMalloc((void **)&dev_Fnorm, sizeof(double) * batch);

#pragma endregion
#pragma region
    hipMalloc((void **)&dev_diag_1,sizeof(double) * minmn);
    hipMalloc((void **)&dev_A_1, sizeof(double) * height * width_perdevice * batch);
    hipMalloc((void **)&dev_V_1, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMalloc((void **)&dev_U_1, sizeof(double) * height * height * batch);
    // hipMalloc((void **)&dev_V0, sizeof(double) * width0 * width0 * batch);

    hipMalloc((void **)&dev_roundRobin_1, sizeof(int) * (2 * k - 1) * 2 * k);

    hipMalloc((void **)&dev_jointG_1, sizeof(double) * 2*k * 2*k * p*batch);
    hipMalloc((void **)&dev_Aij_1, sizeof(double) * height * 2*k * p*batch);

    hipMalloc((void **)&dev_AiAi_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_AiAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_AjAj_1, sizeof(double) * k * k * sliceNum * p1 * batch);
    hipMalloc((void **)&dev_pairsOfEVD_1, sizeof(int) * 2 * p1 * batch);
    hipMalloc((void **)&dev_swap_data_1,sizeof(double)*p1*height*k);

    host_allpass_1 = (unsigned *)malloc(sizeof(unsigned) * batch);
    host_pass_1 = (unsigned *)malloc(sizeof(unsigned) * p1 * batch);
    hipMalloc((void **)&dev_allpass_1, sizeof(unsigned) * batch);
    hipMalloc((void **)&dev_pass_1, sizeof(unsigned) * p1 * batch);

    hipMalloc((void **)&dev_norm_1, sizeof(double) * 2 * p1 * batch);
    hipMalloc((void **)&dev_order_1, sizeof(unsigned int) * 2 * p1 * batch);
    host_Fnorm_1 = (double *)malloc(sizeof(double) * batch);
    hipMalloc((void **)&dev_tempFnorm_1, sizeof(double) * 2 * p1 * batch);
    hipMalloc((void **)&dev_Fnorm_1, sizeof(double) * batch);

#pragma endregion

// preset before svd  
#pragma region

    hipMemset(dev_V, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMemset(dev_U, 0, sizeof(double) * height * height * batch);
    // hipMemset(dev_V0, 0, sizeof(double) * width_perdevice * width_perdevice * batch);
    hipMemset(dev_pairsOfEVD, 0, sizeof(int) * 2 * p * batch); 
    memset(host_pass, 0, sizeof(unsigned) * p * batch);
    hipMemset(dev_pass, 0, sizeof(unsigned) * p * batch);
    int shape[3]={batch,height,width_perdevice};
    double* host_A1,* host_A2;
    host_A1 = (double*)malloc(sizeof(double)*width_perdevice*height);
    host_A2 = (double*)malloc(sizeof(double)*width_perdevice*height);
    double test_result[4] = {0, 1.0, 1.0, 1.0}; // 0:tag, 1:time
    test_result[0] = 2.0;
    hipMemcpy(dev_A,host_A,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice);
    // hipMemcpy(dev_A_1,host_A+height*width_perdevice,sizeof(double)*height*width_perdevice,hipMemcpyHostToDevice);
    // hipMemcpy(host_A2,dev_A_1,sizeof(double)*p1*k*height,hipMemcpyDeviceToHost);
    // // printf("host_A1 %d\n",i);
    // for(int j = 0;j < 2*p1*k*height;++j){
    //     printf("%lf ",host_A[j]);
    // }
    // printf("\n");

    // double* dev_test_A = (double*)malloc(sizeof(double)*2*p*k*height);
    // hipMemcpy(dev_test_A,host_A,sizeof(double)*2*p*k*height,hipMemcpyHostToDevice);
    // int shape1[3] = 
    // our svd
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    clock_t begin = clock();
    dim3 dimGrid0(1, 1, 1);
    dim3 dimBlock0(32, 32, 1);
    hipEventRecord(start, stream);  // 在 stream2 记录开始时间
    nvtxMarkA("Start SVD Computation"); // 这里会在 Nsight Systems 里显示一个标记
    generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream>>>(dev_roundRobin, 2*k);
    svd_large_matrix_1(gpu0,stream,true,dev_A, shape, dev_diag, dev_U, dev_V,dev_V0, th, tw, dev_roundRobin,dev_jointG,dev_Aij,dev_AiAi,dev_AiAj,dev_AjAj,dev_pairsOfEVD,dev_allpass,dev_pass,dev_norm,dev_order,dev_tempFnorm,dev_Fnorm);
    hipEventRecord(stop, stream);  // 在 stream2 记录结束时间

    hipEventSynchronize(stop);  // 确保 stop 事件被正确记录
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    nvtxMarkA("End SVD Computation");
    clock_t end = clock();
    printf("cost:   %lf\n",milliseconds/1000);
    // cusolver svd
    // cusolver_svd(dev_A, shape, dev_diag, dev_U, dev_V, test_result);
    double* host_U = (double*)malloc(sizeof(double)*height*height*batch);
    hipMemcpy(host_U,dev_U,sizeof(double)*height*height*batch,hipMemcpyDeviceToHost);
    FILE* file = fopen("dev_U.txt","w");
    for(int f = 0;f < height;++f){
        for(int g=0;g<height;++g){
            fprintf(file,"%lf ",host_U[f*height+g]);
        }
        fprintf(file,"\n");
    }
    double* host_diag = (double*)malloc(sizeof(double)*minmn*batch);
    // double* host_diag1 = (double*)malloc(sizeof(double)*minmn*batch);
    hipMemcpy(host_diag,dev_diag,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    // hipMemcpy(host_diag1,dev_diag_1,sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
    FILE* file2 = fopen("dev_diag.txt","w");
    for(int f = 0;f < minmn;++f){
        fprintf(file2,"%lf ",host_diag[f]);
    }
    
    printf("matrix:%d×%d×%d, speedup over cusolver: %lf/%lf = %lf\n", batch, height, width, test_result[2], test_result[1], test_result[2]/test_result[1]); 

    free(host_A);
    hipFree(dev_A);
    hipFree(dev_U);
    hipFree(dev_V);
    hipFree(dev_diag);
    hipDeviceReset();
}

int main(int argc, char* argv[]){
    test17();
}