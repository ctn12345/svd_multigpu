#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include<omp.h>
// #include <nvToolsExt.h>
#include "matrix_generate.hpp"
#include "large_matrix_svd.cu"
#include "small_matrix_svd.cu"

// #include "cusolver_svd.cu"

using namespace std;
using namespace std::chrono;



void fill_hostorder_total(int* host_order_total,int** host_order,double** host_norm,int* p,int num_gpus){
    int* idx = (int*)malloc(sizeof(int)*num_gpus);
    memset(idx, 0, sizeof(int) * num_gpus);
    bool flag = false;
    int index = 0;
    while(!flag){
        double tmp = DBL_MAX;
        int need_index = 0;
        for(int u = 0;u < num_gpus;++u){
            if(idx[u] < 2 * p[u] && tmp > host_norm[u][host_order[u][idx[u]]]){
                tmp = host_norm[u][host_order[u][idx[u]]];
                need_index = u;
            }
        }
        
        host_order_total[index] = host_order[need_index][idx[need_index]]+p[0]*2*need_index;
        idx[need_index]++;
        index++;
        flag = true;
        for(int u = 0;u < num_gpus;++u){
            if(idx[u] < 2*p[u]){
                flag = false;
            }
        }
    }
    free(idx);
    // int index=0,idx1=0,idx2=0;
    // while(idx1 < 2*p || idx2 < 2*p1){
    //     if(idx1 < 2*p && idx2 < 2*p1){
    //         if(host_norm_1[host_order_1[idx1]] < host_norm_2[host_order_2[idx2]]){
    //             host_order_total[index]=host_order_1[idx1];
    //             idx1++;
    //             index++;
    //         }
    //         else{
    //             host_order_total[index] = host_order_2[idx2]+2*p;
    //             idx2++;
    //             index++;
    //         }
    //     }
    //     else if(idx1 < 2*p){
    //         host_order_total[index] = host_order_1[idx1];
    //         idx1++;
    //         index++;
    //     }
    //     else{
    //         host_order_total[index] = host_order_2[idx2]+2*p;
    //         idx2++;
    //         index++;
    //     }
    // }
}
// fig 14 a
// 100x512x512 speedup over cusolver(CUDA platform)
void test17(){
    int num_gpus;

    // 获取 GPU 数量
    hipGetDeviceCount(&num_gpus);
    printf("nums gpu is %d\n ",num_gpus);
    int gpu0=0,gpu1=1;
    int batch = 1;
    int height = 64;
    int width = 64;
    int th=0, tw=0;
    // int shape[3] = {batch, height, width};
    int minmn = height > width/num_gpus ? width/num_gpus : height;

    double* host_A = (double*)malloc(sizeof(double) * height * width);
    double* host_V = (double*)malloc(sizeof(double) * width*width);
    string matrix_path1 = "./data/generated_matrixes/A_h" + to_string(height) + "_w" + to_string(width)+ ".txt";

    // read in host A
    FILE* A_fp = fopen(matrix_path1.data(), "r");
    if(A_fp==NULL){
        generate_matrix(height, width);
        A_fp = fopen(matrix_path1.data(), "r");
        if(A_fp==NULL){
            printf("open file falied\n");
            return ;
        }
    }
    for(int i=0; i < height*width; i++){
        fscanf(A_fp, "%lf", &host_A[i]);
    }

    fclose(A_fp);

    tw = 32;
    th = 32;
    int k = tw/2;
    int slice = th;
    int width_perdevice=width/num_gpus;
    size_t pitch;
    int*p_a,*p_b,*p_ab;

    // printf("input matrix shape: %d × %d × %d, tile shape: %d × %d\n", batch, height0, width0, th, tw);

// prams
// definition 1---gpu1
#pragma region
  
    /* p is the count of match-matrix A_ij, 
    e.g. a 16*16 matrix，k=4, 16*8 match-matrix A_ij's count is 2, i.e. p=2. */
    // int p[num_gpus] = {(width_perdevice - 1) / (2 * k) + 1,(width_perdevice - 1) / (2 * k) + 1,(width_perdevice - 1) / (2 * k) + 1,(width_perdevice - 1) / (2 * k) + 1}; 
    // each match-matrix A_ij is cut into slices at column wise, q is the count of these slices 
    // int q[num_gpus] = {(height - 1) / slice + 1,(height - 1) / slice + 1,(height - 1) / slice + 1,(height - 1) / slice + 1};
    // int width_perdevice = p * (2 * k);
    // height = q * slice;
    // 申请指针数组（动态分配）
    int** dev_pa = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pb = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pab = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pa1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pb1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pab1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pa_1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pb_1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pab_1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pa1_1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pb1_1 = (int**)malloc(num_gpus * sizeof(int*));
    int** dev_pab1_1 = (int**)malloc(num_gpus * sizeof(int*));
    double** dev_A = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_V = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_V0 = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_U = (double**)malloc(num_gpus * sizeof(double*));
    int** dev_roundRobin = (int**)malloc(num_gpus * sizeof(int*));
    double** dev_jointG = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_AiAi= (double**)malloc(num_gpus * sizeof(double*));
    double** dev_Aij = (double**)malloc(num_gpus * sizeof(double*)); 
    double** dev_AiAj = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_AjAj = (double**)malloc(num_gpus * sizeof(double*));
    unsigned** dev_allpass = (unsigned**)malloc(num_gpus * sizeof(unsigned*));
    unsigned** dev_pass = (unsigned**)malloc(num_gpus * sizeof(unsigned*));
    double** dev_swap_data = (double**)malloc(num_gpus * sizeof(double*));
    double** dev_norm = (double**)malloc(num_gpus * sizeof(double*));
    unsigned int** dev_order= (unsigned int**)malloc(num_gpus * sizeof(unsigned int*));
    double** dev_tempFnorm= (double**)malloc(num_gpus * sizeof(double*));
    double** dev_Fnorm= (double**)malloc(num_gpus * sizeof(double*));
    double** dev_diag = (double**)malloc(num_gpus * sizeof(double*));
    
    int** dev_pairsOfEVD = (int**)malloc(num_gpus * sizeof(int*));

    int* p = (int*)malloc(num_gpus * sizeof(int));
    int q;
    int sliceNum;
   
    unsigned** host_allpass = (unsigned**)malloc(num_gpus * sizeof(unsigned*));
    unsigned** host_pass = (unsigned**)malloc(num_gpus * sizeof(unsigned*));
    double** value = (double**)malloc(num_gpus * sizeof(double*));
    double** host_Fnorm = (double**)malloc(num_gpus * sizeof(double*));
    double** host_A_per = (double**)malloc(num_gpus * sizeof(double*));
    int** host_order = (int**)malloc(num_gpus * sizeof(int*));
    double** host_rawnorm = (double**)malloc(num_gpus*sizeof(double*));
    double** host_norm = (double**)malloc(num_gpus*sizeof(double*));
    double** host_swap_data = (double**)malloc(sizeof(double*)*num_gpus);
    double** host_swap_V = (double**)malloc(sizeof(double*)*num_gpus);
    double** test_Fnorm = (double**)malloc(sizeof(double*)*num_gpus);
    // int gpuid = 0;
    for(int gpuid = 0;gpuid < num_gpus;++gpuid){
        p[gpuid]=(width_perdevice-1)/(2*k)+1;
        host_order[gpuid] = (int*)malloc(sizeof(int)*p[gpuid]*2*batch);
        host_A_per[gpuid] = (double*)malloc(sizeof(double)*width_perdevice*height);
        
        host_rawnorm[gpuid] = (double*)malloc(sizeof(double)*p[gpuid]*2*batch);
        host_norm[gpuid] = (double*)malloc(sizeof(double)*p[gpuid]*2*batch);
        host_swap_data[gpuid] = (double*)malloc(sizeof(double)*p[gpuid]*height*k);
        host_swap_V[gpuid] = (double*)malloc(sizeof(double)*width_perdevice*width/2);
        test_Fnorm[gpuid] = (double*)malloc(sizeof(double)*batch);
        host_pass[gpuid] = (unsigned*)malloc(sizeof(unsigned)*p[gpuid]*2*batch);
        host_allpass[gpuid] = (unsigned*)malloc(sizeof(unsigned)*batch);
    }
    q = (height-1)/slice+1;
    sliceNum = q;
#pragma endregion
hipError_t err;
for(int gpuid = 0;gpuid < num_gpus;++gpuid){
    hipSetDevice(gpuid);
        
    err = hipMalloc((void**)&dev_pa[gpuid],sizeof(int)*p[gpuid]);
    if (err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(err));
        return;
    }
    hipMalloc((void**)&dev_pb[gpuid],sizeof(int)*p[gpuid]);
    hipMalloc((void**)&dev_pab[gpuid],sizeof(int)*2*p[gpuid]*(2*p[gpuid]-1));
    // next_time
    hipMalloc((void**)&dev_pa1[gpuid],sizeof(int)*p[gpuid]);
    hipMalloc((void**)&dev_pb1[gpuid],sizeof(int)*p[gpuid]);
    hipMalloc((void**)&dev_pab1[gpuid],sizeof(int)*2*p[gpuid]*p[gpuid]);

    hipMalloc((void **)&dev_U[gpuid], sizeof(double) * height * height * batch);
    hipMalloc((void **)&dev_A[gpuid], sizeof(double) * height * width_perdevice * batch);
    err = hipMalloc((void **)&dev_V[gpuid], sizeof(double) * width * width_perdevice * batch);
    if (err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(err));
        return;
    }
    hipMalloc((void **)&dev_V0[gpuid], sizeof(double) * width * width_perdevice * batch);
    hipMalloc((void **)&dev_diag[gpuid],sizeof(double) * minmn);
    hipMalloc((void **)&dev_roundRobin[gpuid], sizeof(int) * (2 * k - 1) * 2 * k);
    hipMalloc((void **)&dev_jointG[gpuid], sizeof(double) * 2*k * 2*k * p[gpuid]*batch);
    hipMalloc((void **)&dev_Aij[gpuid], sizeof(double) * height * 2*k * p[gpuid]*batch);
    hipMalloc((void **)&dev_AiAi[gpuid], sizeof(double) * k * k * sliceNum * p[gpuid] * batch);
    hipMalloc((void **)&dev_AiAj[gpuid], sizeof(double) * k * k * sliceNum * p[gpuid] * batch);
    hipMalloc((void **)&dev_AjAj[gpuid], sizeof(double) * k * k * sliceNum * p[gpuid] * batch);
    hipMalloc((void **)&dev_pairsOfEVD[gpuid], sizeof(int) * 2 * p[gpuid] * batch);
    hipMalloc((void **)&dev_swap_data[gpuid],sizeof(double)*p[gpuid]*height*k);
    hipMalloc((void **)&dev_pass[gpuid],sizeof(unsigned)*p[gpuid]*batch);
    hipMalloc((void **)&dev_norm[gpuid],sizeof(double)*2*p[gpuid]*batch);
    hipMalloc((void **)&dev_Fnorm[gpuid], sizeof(double) * batch);
    hipMalloc((void **)&dev_tempFnorm[gpuid], sizeof(double) * 2*p[gpuid]*batch);
    hipMalloc((void **)&dev_order[gpuid],sizeof(unsigned int)*2*p[gpuid]*batch);
    hipMalloc((void **)&dev_allpass[gpuid],sizeof(unsigned)*batch);
    err = hipMemset(dev_V[gpuid], 0,  sizeof(double)*width * width_perdevice * batch);
    if (err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(err));
        return;
    }
    hipMemset(dev_U[gpuid], 0,  sizeof(double)*height * height * batch);
    hipMemset(dev_diag[gpuid], 0,  sizeof(double)*minmn);
    hipMemset(dev_V0[gpuid], 0, sizeof(double) * width * width_perdevice * batch);
    hipMemset(dev_pairsOfEVD[gpuid], 0,  sizeof(int) *2 * p[gpuid] * batch); 
    hipMemset(dev_pass[gpuid], 0,  sizeof(unsigned)*p[gpuid] * batch);
}
    int shape[3]={batch,height,width_perdevice};
    double test_result[4] = {0, 1.0, 1.0, 1.0}; // 0:tag, 1:time
    test_result[0] = 2.0;

    hipStream_t* stream = (hipStream_t*)malloc(num_gpus * sizeof(hipStream_t));
    for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipStreamCreate(&stream[i]);
    }
    for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipMemcpyAsync(dev_A[i],host_A+i*width_perdevice*height,sizeof(double)*width_perdevice*height,hipMemcpyHostToDevice,stream[i]);
    }
    omp_set_num_threads(num_gpus);
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        Multi_init_dev_V<<<batch,256,0,stream[gpuid]>>>(dev_V[gpuid],width_perdevice,width,gpuid);
    }
    clock_t c1 = clock();
    dim3 dimGrid0(1, 1, 1);
    dim3 dimBlock0(32, 32, 1);
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        generate_roundRobin_128<<<dimGrid0, dimBlock0,0,stream[gpuid]>>>(dev_roundRobin[gpuid], 2*k);
    }
    
    // printf("dev_V \n");
    // hipSetDevice(0);
    // double* test_V = (double*)malloc(sizeof(double)*width*width_perdevice);
    // hipMemcpyAsync(test_V,dev_V[0],sizeof(double)*width*width_perdevice,hipMemcpyDeviceToHost,stream[0]);
    // for(int i = 0;i < width_perdevice;++i){
    //    for(int g = 0;g < width;++g){
    //        printf("%f ",test_V[i*width+g]);
    //    }
    //    printf("\n");
    // }
    // printf("dev_1\n");
    // hipSetDevice(1);
    // hipMemcpyAsync(test_V,dev_V[1],sizeof(double)*width*width_perdevice,hipMemcpyDeviceToHost,stream[1]);
    // for(int i = 0;i < width_perdevice;++i){
    //    for(int g = 0;g < width;++g){
    //     if(test_V[i*width+g] == 1){
    //         printf("%d %d ",i,g);
    //         printf("\n");
    //     }
        
    //     //    printf("%f ",test_V[i*width+g]);
    //    }
    //    printf("\n");
    // }
    bool continue_flag = false;

    // omp_set_num_threads(num_gpus);
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        getRankNewNew_2<<<1,1024,0,stream[gpuid]>>>(2*p[gpuid],dev_pab[gpuid],dev_pa[gpuid],dev_pb[gpuid]);
        getRankNewNew_1<<<1,1024,0,stream[gpuid]>>>(p[gpuid],dev_pa1[gpuid],dev_pb1[gpuid],dev_pab1[gpuid]);
    }
    // printf("pab %d \n",p[0]);

    // int* test_pab=(int*)malloc(sizeof(int)*2*p[0]*(2*p[0]-1));
    // hipMemcpyAsync(test_pab,dev_pab[0],sizeof(int)*(2*p[0]-1)*2*p[0],hipMemcpyDeviceToHost,stream[0]);
    // for(int i = 0;i < (2*p[0]-1)*2*p[0];++i){
    //     printf("%d ",test_pab[i]);
    // }
    // printf("\n");
    int sweep = 0,maxsweep = 11;
    double svd_tol = 1e-7;
    int* raw_host_order = (int*)malloc(sizeof(int)*2*p[0]*num_gpus*batch);
    hipStreamSynchronize(stream[0]);
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        compute_norm<<<2 * p[gpuid] * batch, 128,0,stream[gpuid]>>>(dev_A[gpuid], dev_norm[gpuid], dev_order[gpuid], height, width_perdevice, p[gpuid], q, k);
        binoticSort_original<<<batch, 1024,0,stream[gpuid]>>>(dev_norm[gpuid], dev_order[gpuid], 2 * p[gpuid], p[gpuid]);
        
    } 
    // hipDeviceSynchronize();
    for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipMemcpyAsync(host_order[i],dev_order[i],sizeof(int)*2*p[i]*batch,hipMemcpyDeviceToHost,stream[i]);
        hipMemcpyAsync(host_norm[i],dev_norm[i],sizeof(double)*2*p[i]*batch,hipMemcpyDeviceToHost,stream[i]);
    }
    for(int i = 0;i < num_gpus;++i){
        for(int j = 0;j < 2*p[0];++j){
            host_rawnorm[i][host_order[i][j]] = host_norm[i][j];
        }
    }
    // printf("host norm \n");
    // for(int g = 0;g < 2*p[0];++g){
    //     printf("%f ",host_norm[0][g]);
    // }
    // printf("\n");
    int* host_index=(int*)malloc(sizeof(int)*2*p[0]*num_gpus);
    // int* host_pass = (int*)malloc(sizeof(int)*10);
    fill_hostorder_total(raw_host_order,host_order,host_rawnorm,p,num_gpus);
    // test raw_host_order part
    // for(int g = 0;g < num_gpus*2*p[0];++g){
    //     int gpuid = raw_host_order[g] / (2*p[0]);
    //     printf("%f ",host_rawnorm[gpuid][raw_host_order[g]-gpuid * 2*p[0]]);
    // }
    // printf("\n");
    for(int i = 0;i < num_gpus*2*p[0];++i){
        host_index[raw_host_order[i]] = i;
    }
    #pragma omp parallel
    {
        int gpuid = omp_get_thread_num();
        hipSetDevice(gpuid);
        if(height >= 32){
            computeFnorm1<<<2 * p[gpuid] * batch, 128,0,stream[gpuid]>>>(dev_A[gpuid], dev_tempFnorm[gpuid], p[gpuid], height/32, height, width_perdevice, k);
        }
        else{
            computeFnorm1<<<2 * p[gpuid] * batch, 128,0,stream[gpuid]>>>(dev_A[gpuid], dev_tempFnorm[gpuid], p[gpuid], 1, height, width_perdevice, k);   
        }
        // hipDeviceSynchronize();
        computeFnorm2<<<batch, 32,0,stream[gpuid]>>>(dev_tempFnorm[gpuid], dev_Fnorm[gpuid], p[gpuid]);  //&1.3
    }
    for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipMemcpy(test_Fnorm[i],dev_Fnorm[i],sizeof(double)*batch,hipMemcpyDeviceToHost);
    }
    double* Fin_Fnorm = (double*)malloc(sizeof(double)*batch);
    for(int bat = 0;bat < batch;++bat){
        for(int i = 0;i < num_gpus;++i)
            Fin_Fnorm[bat] += test_Fnorm[i][bat];
    }
    for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipMemcpyAsync(dev_Fnorm[i],Fin_Fnorm,sizeof(double)*batch,hipMemcpyHostToDevice,stream[i]);
    }
    // before have been tested

    float elapsed_time = 0;
    float milliseconds = 0;
    int* host_order_total = (int*)malloc(sizeof(int)*2*p[0]*num_gpus);
    printf("FIN Form %f \n",Fin_Fnorm[0]);
    double* host_jointG = (double*)malloc(sizeof(double)*2*k*2*k);
    double* test_A = (double*)malloc(sizeof(double)*p[0]*k*height);
    double* test_aij = (double*)malloc(sizeof(double)*k);
    hipError_t errf;
    while(!continue_flag){ 
        // part1
        dim3 dimGrid77(sliceNum, p[0], batch);// 2×2×100个block，每个block 256线程
        dim3 dimGrid7(p[0], batch, 1);
        // printf("EVD 1\n");
        // int times = 1;
        omp_set_num_threads(num_gpus);
        
        // hipMemcpy(test_A,dev_A[0],sizeof(double)*p[0]*k*height,hipMemcpyDeviceToHost);
        // printf("devA\n");
        // for(int g = 0;g < p[0];++g){
        //     for(int f = 0;f < 5;++f)
        //         printf("%f ",test_A[g*k*height + f]);
        //     printf("\n");
        // }
        // printf("\n");
       
        // printf("EVD start \n");
        for(int i = 0;i < 2*p[0]-1;++i){
            #pragma omp parallel
            {
                int gpuid = omp_get_thread_num();
                hipSetDevice(gpuid);
                generate_jointG00_1<<<dimGrid77, 256,0,stream[gpuid]>>>(dev_pab[gpuid],dev_A[gpuid], height, width_perdevice, p[gpuid], q, dev_pairsOfEVD[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid],i,  k, slice, sliceNum);    //&1.3
               
                generate_jointG21<<<dimGrid7, 256,0,stream[gpuid]>>>(dev_jointG[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid], dev_Fnorm[gpuid], dev_pass[gpuid], p[gpuid], k, sliceNum, svd_tol);    //&1.3
                MUL_EVD_1(stream[gpuid],dev_jointG[gpuid], dev_A[gpuid], dev_V[gpuid], dev_pairsOfEVD[gpuid], p[gpuid], q, height,width, width_perdevice, dev_roundRobin[gpuid], batch, k, slice, sliceNum, sweep); //&1.3
            }    
        }
        //test 1
        // printf("\nEVD1\n");
        
        // hipMemcpy(host_jointG,dev_jointG[0],sizeof(double)*4*k*k,hipMemcpyDeviceToHost);
        // for(int i = 0;i < 10;++i){
        //     printf("%f ",host_jointG[i]);
        // }
        // printf("\n");

        // break;

        int init_time = 1;
        while(init_time < num_gpus){
            for(int total_time = 1;total_time < num_gpus / init_time;++total_time){
                for(int i = 0;i<num_gpus;++i){
                    hipSetDevice(i);
                    hipMemcpyAsync(host_swap_data[i],dev_A[i]+p[i]*k*height,sizeof(double)*p[i]*k*height,hipMemcpyDeviceToHost,stream[i]);
                }
                for(int per_time = 1;per_time <= init_time;++per_time){
                    for(int i = num_gpus/init_time * (per_time-1);i < num_gpus/init_time * per_time;++i){
                        hipSetDevice(i);
                        if(i == num_gpus/init_time * per_time-1){
                            hipMemcpyAsync(dev_A[i]+p[i]*k*height,host_swap_data[num_gpus/init_time*(per_time-1)],sizeof(double)*p[i]*k*height,hipMemcpyHostToDevice,stream[i]);
                            hipMemcpyAsync(dev_V[i]+width*width_perdevice/2,host_swap_V[num_gpus/init_time*(per_time-1)],sizeof(double)*width*width_perdevice/2,hipMemcpyHostToDevice,stream[i]);
                        }
                        else{
                            hipMemcpyAsync(dev_A[i]+p[i]*k*height,host_swap_data[i+1],sizeof(double)*p[i]*k*height,hipMemcpyHostToDevice,stream[i]);
                            hipMemcpyAsync(dev_V[i]+width*width_perdevice/2,host_swap_V[i+1],sizeof(double)*width*width_perdevice/2,hipMemcpyHostToDevice,stream[i]);
                        }    
                    }
                }
                for(int i = 0;i < p[0];++i){
                    #pragma omp parallel
                    {
                        int gpuid = omp_get_thread_num();
                        hipSetDevice(gpuid);
                        generate_jointG00_1<<<dimGrid77, 256,0,stream[gpuid]>>>(dev_pab[gpuid],dev_A[gpuid], height, width_perdevice, p[gpuid], q, dev_pairsOfEVD[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid],i,  k, slice, sliceNum);    //&1.3
                        generate_jointG21<<<dimGrid7, 256,0,stream[gpuid]>>>(dev_jointG[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid], dev_Fnorm[gpuid], dev_pass[gpuid], p[gpuid], k, sliceNum, svd_tol);    //&1.3
                        MUL_EVD_1(stream[gpuid],dev_jointG[gpuid], dev_A[gpuid], dev_V[gpuid], dev_pairsOfEVD[gpuid], p[gpuid], q, height, width,width_perdevice, dev_roundRobin[gpuid], batch, k, slice, sliceNum, sweep); //&1.3
                    }    
                }  
                
            }
            // refresh new part
            init_time *= 2;
            if(init_time <= num_gpus){
                for(int i = 1;i <= init_time;++i){
                    int diff = i%2==1?p[0]*k*height:0;
                    int diff_V = i%2 == 1?width_perdevice*width/2:0;
                    for(int g = num_gpus/init_time*(i-1);g < num_gpus/init_time * i;++g){
                        hipSetDevice(g);
                        hipMemcpyAsync(host_swap_data[g],dev_A[g]+diff,sizeof(double)*p[g]*k*height,hipMemcpyDeviceToHost,stream[g]);
                        hipMemcpyAsync(host_swap_V[g],dev_V[g]+diff_V,sizeof(double)*width_perdevice*width/2,hipMemcpyDeviceToHost,stream[g]);
                    }        
                }
                for(int i = 1;i <= init_time;++i){
                    int diff = i%2==1?p[0]*k*height:0;
                    int diff_V = i%2 == 1?width_perdevice*width/2:0;
                    int flag = i%2;
                    for(int g = num_gpus/init_time*(i-1);g < num_gpus/init_time * i;++g){
                        hipSetDevice(g);
                        if(flag == 0){
                            hipMemcpyAsync(dev_A[g]+diff,host_swap_data[g-num_gpus/init_time],sizeof(double)*p[g]*k*height,hipMemcpyHostToDevice,stream[g]);
                            hipMemcpyAsync(dev_V[i]+diff_V,host_swap_V[g-num_gpus/init_time],sizeof(double)*width*width_perdevice/2,hipMemcpyHostToDevice,stream[g]);
                        }
                        else{
                            hipMemcpyAsync(dev_A[g]+diff,host_swap_data[g+num_gpus/init_time],sizeof(double)*p[g]*k*height,hipMemcpyHostToDevice,stream[g]);
                            hipMemcpyAsync(dev_V[i]+diff_V,host_swap_V[g+num_gpus/init_time],sizeof(double)*width*width_perdevice/2,hipMemcpyHostToDevice,stream[g]);
                        }
                        }
                           
                    }        
                
                for(int i = 0;i < p[0];++i){
                    #pragma omp parallel
                    {
                        int gpuid = omp_get_thread_num();
                        hipSetDevice(gpuid);
                        generate_jointG00_1<<<dimGrid77, 256,0,stream[gpuid]>>>(dev_pab[gpuid],dev_A[gpuid], height, width_perdevice, p[gpuid], q, dev_pairsOfEVD[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid],i,  k, slice, sliceNum);    //&1.3
                        generate_jointG21<<<dimGrid7, 256,0,stream[gpuid]>>>(dev_jointG[gpuid], dev_AiAi[gpuid], dev_AiAj[gpuid], dev_AjAj[gpuid], dev_Fnorm[gpuid], dev_pass[gpuid], p[gpuid], k, sliceNum, svd_tol);    //&1.3
                        MUL_EVD_1(stream[gpuid],dev_jointG[gpuid], dev_A[gpuid], dev_V[gpuid], dev_pairsOfEVD[gpuid], p[gpuid], q, height, width,width_perdevice, dev_roundRobin[gpuid], batch, k, slice, sliceNum, sweep); //&1.3
                    }    
                }  
            }      
        }
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            hipSetDevice(gpuid);
            // printf("gpuid %d \n",gpuid);
            compute_norm<<<2 * p[gpuid] * batch, 128,0,stream[gpuid]>>>(dev_A[gpuid], dev_norm[gpuid], dev_order[gpuid], height, width_perdevice, p[gpuid], q, k);
            binoticSort_original<<<batch, 1024,0,stream[gpuid]>>>(dev_norm[gpuid], dev_order[gpuid], 2 * p[gpuid], p[gpuid]);
        } 
        for(int i = 0;i < num_gpus;++i){
            hipSetDevice(i);
            hipMemcpyAsync(host_order[i],dev_order[i],sizeof(int)*2*p[i]*batch,hipMemcpyDeviceToHost,stream[i]);
            hipMemcpyAsync(host_norm[i],dev_norm[i],sizeof(double)*2*p[i]*batch,hipMemcpyDeviceToHost,stream[i]);
        }
        for(int i = 0;i < num_gpus;++i){
            for(int j = 0;j < 2*p[0];++j){
                host_rawnorm[i][host_order[i][j]] = host_norm[i][j];
            }
        }

        fill_hostorder_total(host_order_total,host_order,host_rawnorm,p,num_gpus);
        // printf("host order total\n");
        for(int g = 0;g < num_gpus*p[0]*2;++g){
            // printf("%d ",host_order_total[g]);
            int gpuid = host_order_total[g] / (2*p[0]);
            int index = host_order_total[g] % (2*p[0]);
            // printf("%f ",host_rawnorm[gpuid][index]);
        }
        // printf("\n");
        // hipMemcpy(host_pass[0],dev_pass[0],sizeof(int)*p[0],hipMemcpyDeviceToHost);
        // printf("host pass\n");
        // for(int g = 0;g < p[0];++g){
        //     printf("%d ",host_pass[0][0]);
        // }
        // printf("\n");
        for(int i = 0;i < num_gpus;++i){
            hipSetDevice(i);
            hipMemcpyAsync(host_A+i*width_perdevice*height,dev_A[i],sizeof(double)*width_perdevice*height,hipMemcpyDeviceToHost,stream[i]);
            hipMemcpyAsync(host_V+i*width_perdevice*width,dev_V[i],sizeof(double)*width_perdevice*width,hipMemcpyDeviceToHost,stream[i]);
        }
        int cnt = 0;
        for(int i = 0;i<num_gpus;++i){
            hipSetDevice(i);
            int per_len = 0;
            for(int index = i*2*p[0];index < (i+1)*2*p[0];++index){
                hipMemcpyAsync(dev_A[i]+per_len*k*height,&host_A[host_order_total[host_index[cnt]]*k*height],sizeof(double)*k*height,hipMemcpyHostToDevice,stream[i]);
                hipMemcpyAsync(dev_V[i]+per_len*k*width,&host_V[host_order_total[host_index[cnt]]*k*width],sizeof(double)*k*width,hipMemcpyHostToDevice,stream[i]);
                ++cnt;
                per_len++;
            }
        }
        ++sweep;
        #pragma omp parallel
        {
            int gpuid = omp_get_thread_num();
            hipSetDevice(gpuid);
            judgeFunc<<<batch, 1024,0,stream[gpuid]>>>(dev_allpass[gpuid], dev_pass[gpuid], p[gpuid]);   // concentrate each block's result(converged or not)
        }
        bool tempFlag = true;
        for(int i = 0;i < num_gpus;++i){
            hipMemcpy(host_allpass[i],dev_allpass[i],sizeof(unsigned)*batch,hipMemcpyDeviceToHost);
        }
        // printf("host pass\n");
        for(int i = 0;i < num_gpus;++i){
            // printf("%d ",host_allpass[i][0]);
            if(!ifallpass(host_allpass[i],batch,p[i])){
                tempFlag = false;
            }
        }
        // printf("\n");
        continue_flag = (tempFlag || sweep>maxsweep);
        for(int g = 0;g < num_gpus;++g){
            hipSetDevice(g);
            hipStreamSynchronize(stream[g]);
        }
    }
     #pragma omp parallel
     {
        //  int gpuid = omp_get_thread_num();
         
         int gpuid = omp_get_thread_num();
         hipSetDevice(gpuid);
         

         dim3 dimGrid10(2 * p[gpuid], batch, 1);
         dim3 dimBlock10(32, k, 1);
         getUDV<<<dimGrid10, dimBlock10,0,stream[gpuid]>>>(dev_A[gpuid], dev_U[gpuid], dev_V[gpuid], dev_V0[gpuid], height, width_perdevice, height, width_perdevice, p[gpuid], height/32, dev_diag[gpuid], width_perdevice, k);  //&1.3
        //  gpuid = 1;
        //  hipSetDevice(gpuid);
        //  // int gpuid = omp_get_thread_num();
        //  getUDV<<<dimGrid10, dimBlock10,0,stream[gpuid]>>>(dev_A[gpuid], dev_U[gpuid], dev_V[gpuid], dev_V0[gpuid], height, width_perdevice, height, width_perdevice, p[gpuid], height/32, dev_diag[gpuid], width_perdevice, k);  //&1.3
     }
     clock_t c2 = clock();
     printf("it costs %f s\n",(double)(c2-c1)/CLOCKS_PER_SEC); 
    //  hipError_t err1 = hipGetLastError();
    // if (err1 != hipSuccess) {
    //     printf("CUDA Error: %s\n", hipGetErrorString(err1));
    // }
     printf("sweep:%d \n",sweep);
     double** host_diag = (double**)malloc(sizeof(double*)*num_gpus);
     for(int i = 0;i < num_gpus;++i){
         host_diag[i] = (double*)malloc(sizeof(double)*minmn*batch);
     }
     FILE* file2 = fopen("dev_diag.txt","w");
    for(int i = 0;i < num_gpus;++i){
        hipMemcpy(host_diag[i],dev_diag[i],sizeof(double)*minmn*batch,hipMemcpyDeviceToHost);
     }
     for(int i = 0;i < num_gpus;++i){
         for(int g = 0;g<minmn;++g)
         fprintf(file2,"%lf ",host_diag[i][g]);
     }

     for(int i = 0;i < num_gpus;++i){
        hipSetDevice(i);
        hipMemcpyAsync(host_V+i*width*width_perdevice,dev_V[i],sizeof(double)*width_perdevice*width,hipMemcpyDeviceToHost,stream[i]);
     }
    FILE* file_V = fopen("dev_V.txt","w");
    for(int i  = 0;i < width;++i){
        for(int j =0;j < width;++j){
            fprintf(file_V,"%lf ",host_V[i*width+j]);
        }
        fprintf(file_V,"\n");
    }
    // for(int i = 0;i < num_gpus;++i){
    //     hipSetDevice(i);
    //     hipFree(dev_A[i]);
    //     hipFree(dev_U[i]);
    //     hipFree(dev_V[i]);
    //     hipFree(dev_V0[i]);
    //     hipStreamDestroy(stream[i]);
    // }
    // free(host_A);
}

int main(int argc, char* argv[]){
    test17();
    // hipblasDestroy(handle);
    // hipDeviceReset();
}